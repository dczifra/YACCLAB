#include "hip/hip_runtime.h"
// Copyright (c) 2020, the YACCLAB contributors, as 
// shown by the AUTHORS file. All rights reserved.
//
// Use of this source code is governed by a BSD-style
// license that can be found in the LICENSE file.

#include <opencv2/cudafeatures2d.hpp>

#include "hip/hip_runtime.h"
#include ""

#include "labeling_algorithms.h"
#include "register.h"

// Oliveira2010

#define BLOCK_ROWS 16
#define BLOCK_COLS 16

using namespace cv;

namespace {

	// Returns the root index of the UFTree
	__device__ unsigned Find(const int *s_buf, unsigned n) {
		// Warning: do not call Find on a background pixel

		unsigned label = s_buf[n];

		assert(label > 0);

		while (label - 1 != n) {
			n = label - 1;
			label = s_buf[n];

			assert(label > 0);
		}

		return n;

	}


	// Merges the UFTrees of a and b, linking one root to the other
	__device__ void Union(int *s_buf, unsigned a, unsigned b) {

		bool done;

		do {

			a = Find(s_buf, a);
			b = Find(s_buf, b);

			if (a < b) {
				int old = atomicMin(s_buf + b, a + 1);
				done = (old == b + 1);
				b = old - 1;
			}
			else if (b < a) {
				int old = atomicMin(s_buf + a, b + 1);
				done = (old == a + 1);
				a = old - 1;
			}
			else {
				done = true;
			}

		} while (!done);

	}


	__global__ void LocalMerge(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

		unsigned local_row = threadIdx.y;
		unsigned local_col = threadIdx.x;
		unsigned local_index = local_row * blockDim.x + local_col;

		unsigned global_row = blockIdx.y * BLOCK_ROWS + local_row;
		unsigned global_col = blockIdx.x * blockDim.x + local_col;
		unsigned img_index = global_row * img.step + global_col;

		__shared__ int s_buf[BLOCK_ROWS * BLOCK_COLS];
		__shared__ unsigned char s_img[BLOCK_ROWS * BLOCK_COLS];

		bool in_limits = (global_row < img.rows && global_col < img.cols);

		s_buf[local_index] = local_index + 1;
		s_img[local_index] = in_limits ? img[img_index] : 0xFF;

		__syncthreads();

		unsigned char v = s_img[local_index];

		if (in_limits) {

			if (v) {

				if (local_col > 0 && s_img[local_index - 1]) {
					Union(s_buf, local_index, local_index - 1);
				}


				if (local_row > 0 && s_img[local_index - blockDim.x]) {
					Union(s_buf, local_index, local_index - blockDim.x);
				}

			}

			else {
				if (local_row > 0 && s_img[local_index - blockDim.x]) {

					if (local_col > 0 && s_img[local_index - 1]) {
						Union(s_buf, local_index - 1, local_index - blockDim.x);
					}


					if (local_col < blockDim.x - 1 && s_img[local_index + 1]) {
						Union(s_buf, local_index + 1, local_index - blockDim.x);
					}
				}

			}

		}

		__syncthreads();

		if (in_limits) {

			if (v) {
				unsigned f = Find(s_buf, local_index);
				unsigned f_row = f / blockDim.x;
				unsigned f_col = f % blockDim.x;
				unsigned global_f = (blockIdx.y * BLOCK_ROWS + f_row) * (labels.step / labels.elem_size) + (blockIdx.x * blockDim.x + f_col);
				labels.data[global_row * labels.step / sizeof(int) + global_col] = global_f + 1;		// C'� distinzione tra background e foreground
			}

			else {
				labels.data[global_row * labels.step / sizeof(int) + global_col] = 0;
			}

		}
	}


	__global__ void LocalMergeSize(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

		unsigned local_row = threadIdx.y;
		unsigned local_col = threadIdx.x;
		unsigned local_index = local_row * blockDim.x + local_col;

		unsigned global_row = blockIdx.y * blockDim.y + local_row;
		unsigned global_col = blockIdx.x * blockDim.x + local_col;
		unsigned img_index = global_row * img.step + global_col;

		extern __shared__ unsigned char shared_memory[];

		int* s_buf = reinterpret_cast<int*>(shared_memory);
		unsigned char* s_img = reinterpret_cast<unsigned char*>(shared_memory + blockDim.y * blockDim.x * sizeof(int));

		bool in_limits = (global_row < img.rows&& global_col < img.cols);

		s_buf[local_index] = local_index + 1;
		s_img[local_index] = in_limits ? img[img_index] : 0xFF;

		__syncthreads();

		unsigned char v = s_img[local_index];

		if (in_limits) {

			if (v) {

				if (local_col > 0 && s_img[local_index - 1]) {
					Union(s_buf, local_index, local_index - 1);
				}


				if (local_row > 0 && s_img[local_index - blockDim.x]) {
					Union(s_buf, local_index, local_index - blockDim.x);
				}

			}

			else {
				if (local_row > 0 && s_img[local_index - blockDim.x]) {

					if (local_col > 0 && s_img[local_index - 1]) {
						Union(s_buf, local_index - 1, local_index - blockDim.x);
					}


					if (local_col < blockDim.x - 1 && s_img[local_index + 1]) {
						Union(s_buf, local_index + 1, local_index - blockDim.x);
					}
				}

			}

		}

		__syncthreads();

		if (in_limits) {

			if (v) {
				unsigned f = Find(s_buf, local_index);
				unsigned f_row = f / blockDim.x;
				unsigned f_col = f % blockDim.x;
				unsigned global_f = (blockIdx.y * blockDim.y + f_row) * (labels.step / labels.elem_size) + (blockIdx.x * blockDim.x + f_col);
				labels.data[global_row * labels.step / sizeof(int) + global_col] = global_f + 1;		// C'è distinzione tra background e foreground
			}

			else {
				labels.data[global_row * labels.step / sizeof(int) + global_col] = 0;
			}

		}
	}



	__global__ void GlobalMerge(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

		unsigned local_row = threadIdx.y;
		unsigned local_col = threadIdx.x;

		unsigned global_row = blockIdx.y * blockDim.y + local_row;
		unsigned global_col = blockIdx.x * blockDim.x + local_col;
		unsigned img_index = global_row * img.step + global_col;
		unsigned labels_index = global_row * (labels.step / labels.elem_size) + global_col;

		bool in_limits = (global_row < img.rows && global_col < img.cols);

		if (in_limits) {

			unsigned char v = img[img_index];

			if (v) {

				if (global_col > 0 && local_col == 0 && img[img_index - 1]) {
					Union(labels.data, labels_index, labels_index - 1);
				}

				if (global_row > 0 && local_row == 0 && img[img_index - img.step]) {
					Union(labels.data, labels_index, labels_index - labels.step / sizeof(int));
				}

			}

			else {

				if (global_row > 0 && img[img_index - img.step]) {

					if (global_col > 0 && (local_row == 0 || local_col == 0) && img[img_index - 1]) {
						Union(labels.data, labels_index - 1, labels_index - labels.step / sizeof(int));
					}

					if ((global_col < img.cols - 1) && (local_row == 0 || local_col == blockDim.x - 1) && img[img_index + 1]) {
						Union(labels.data, labels_index + 1, labels_index - labels.step / sizeof(int));
					}
				}
			}

		}
	}


	__global__ void PathCompression(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

		unsigned global_row = blockIdx.y * blockDim.y + threadIdx.y;
		unsigned global_col = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned labels_index = global_row * (labels.step / labels.elem_size) + global_col;

		if (global_row < labels.rows && global_col < labels.cols) {
			unsigned char val = img[global_row * img.step + global_col];
			if (val) {
				labels[labels_index] = Find(labels.data, labels_index) + 1;
			}
		}
	}

}

class UF : public GpuLabeling2D<Connectivity2D::CONN_8> {
private:
	dim3 grid_size_;
	dim3 block_size_;

public:
	UF() {}

	void PerformLabeling() {

		d_img_labels_.create(d_img_.size(), CV_32SC1);
		grid_size_ = dim3((d_img_.cols + BLOCK_COLS - 1) / BLOCK_COLS, (d_img_.rows + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
		block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);

		// Phase 1
		// CCL on tiles		
		LocalMerge << <grid_size_, block_size_ >> >(d_img_, d_img_labels_);

		// Immagine di debug della prima fase
		//cuda::GpuMat d_local_labels;
		//d_img_labels_.copyTo(d_local_labels);
		//PathCompression << <grid_size_, block_size_ >> > (d_img_, d_local_labels);
		//// ZeroBackground << <grid_size_, block_size_ >> > (d_img_, d_local_labels);
		//Mat1i local_labels(img_.size());
		//d_local_labels.download(local_labels);

		// Phase 2
		// Merges UFTrees of different tiles
		GlobalMerge << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);

		// Immagine di debug della seconda fase
		//cuda::GpuMat d_global_labels;
		//d_img_labels_.copyTo(d_global_labels);
		//PathCompression << <grid_size_, block_size_ >> > (d_img_, d_global_labels);
		//// ZeroBackground << <grid_size_, block_size_ >> > (d_img_, d_global_labels);
		//Mat1i global_labels(img_.size());
		//d_global_labels.download(global_labels);

		// Phase 3
		// Collapse UFTrees
		PathCompression << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);

		hipDeviceSynchronize();
	}


	void PerformLabelingBlocksize(int x, int y, int z) override {

		const int block_cols = x;
		const int block_rows = y;

		d_img_labels_.create(d_img_.size(), CV_32SC1);
		grid_size_ = dim3((d_img_.cols + block_cols - 1) / block_cols, (d_img_.rows + block_rows - 1) / block_rows, 1);
		block_size_ = dim3(block_cols, block_rows, 1);
		int shared_size = block_rows * block_cols * (sizeof(int) + sizeof(unsigned char));

		BLOCKSIZE_KERNEL(LocalMergeSize, grid_size_, block_size_, shared_size, d_img_, d_img_labels_)

		BLOCKSIZE_KERNEL(GlobalMerge, grid_size_, block_size_, 0, d_img_, d_img_labels_)

		BLOCKSIZE_KERNEL(PathCompression, grid_size_, block_size_, 0, d_img_, d_img_labels_)
	}


private:
	double Alloc() {
		perf_.start();
		d_img_labels_.create(d_img_.size(), CV_32SC1);
		perf_.stop();
		return perf_.last();
	}

	double Dealloc() {
		perf_.start();
		perf_.stop();
		return perf_.last();
	}

	double MemoryTransferHostToDevice() {
		perf_.start();
		d_img_.upload(img_);
		perf_.stop();
		return perf_.last();
	}

	void MemoryTransferDeviceToHost() {
		d_img_labels_.download(img_labels_);
	}

	void LocalScan() {
		grid_size_ = dim3((d_img_.cols + BLOCK_COLS - 1) / BLOCK_COLS, (d_img_.rows + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
		block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);
		LocalMerge << <grid_size_, block_size_ >> >(d_img_, d_img_labels_);
		hipDeviceSynchronize();
	}

	void GlobalScan() {
		GlobalMerge << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);
		PathCompression << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);
		hipDeviceSynchronize();
	}

public:
	void PerformLabelingWithSteps()
	{
		double alloc_timing = Alloc();

		perf_.start();
		LocalScan();
		perf_.stop();
		perf_.store(Step(StepType::FIRST_SCAN), perf_.last());

		perf_.start();
		GlobalScan();
		perf_.stop();
		perf_.store(Step(StepType::SECOND_SCAN), perf_.last());

		double dealloc_timing = Dealloc();

		perf_.store(Step(StepType::ALLOC_DEALLOC), alloc_timing + dealloc_timing);

	}

};

REGISTER_LABELING(UF);

REGISTER_KERNELS(UF, LocalMergeSize, GlobalMerge, PathCompression)