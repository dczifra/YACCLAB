#include "hip/hip_runtime.h"
// Copyright (c) 2020, the YACCLAB contributors, as 
// shown by the AUTHORS file. All rights reserved.
//
// Use of this source code is governed by a BSD-style
// license that can be found in the LICENSE file.

#include <opencv2/cudafeatures2d.hpp>

#include "hip/hip_runtime.h"
#include ""

#include "labeling_algorithms.h"
#include "register.h"


#define BLOCK_ROWS 16
#define BLOCK_COLS 16

using namespace cv;

namespace {

	// Only use it with unsigned numeric types
	template <typename T>
	__device__ __forceinline__ unsigned char HasBit(T bitmap, unsigned char pos) {
		return (bitmap >> pos) & 1;
	}

	//__device__ __forceinline__ void SetBit(unsigned char &bitmap, unsigned char pos) {
	//	bitmap |= (1 << pos);
	//}

	// Returns the root index of the UFTree
	__device__ unsigned Find(const int *s_buf, unsigned n) {
		while (s_buf[n] != n) {
			n = s_buf[n];
		}
		return n;
	}


	// Merges the UFTrees of a and b, linking one root to the other
	__device__ void Union(int *s_buf, unsigned a, unsigned b) {

		bool done;

		do {

			a = Find(s_buf, a);
			b = Find(s_buf, b);

			if (a < b) {
				int old = atomicMin(s_buf + b, a);
				done = (old == b);
				b = old;
			}
			else if (b < a) {
				int old = atomicMin(s_buf + a, b);
				done = (old == a);
				a = old;
			}
			else {
				done = true;
			}

		} while (!done);

	}


	__global__ void InitLabeling(cuda::PtrStepSzi labels) {
		unsigned row = (blockIdx.y * BLOCK_ROWS + threadIdx.y) * 2;
		unsigned col = (blockIdx.x * BLOCK_COLS + threadIdx.x) * 2;
		unsigned labels_index = row * (labels.step / labels.elem_size) + col;

		if (row < labels.rows && col < labels.cols) {
			labels[labels_index] = labels_index;
		}
	}

	__global__ void Merge(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

		unsigned row = (blockIdx.y * BLOCK_ROWS + threadIdx.y) * 2;
		unsigned col = (blockIdx.x * BLOCK_COLS + threadIdx.x) * 2;
		unsigned img_index = row * img.step + col;
		unsigned labels_index = row * (labels.step / labels.elem_size) + col;

		if (row < labels.rows && col < labels.cols) {

#define CONDITION_B col>0 && row>1 && img.data[img_index - 2 * img.step - 1]
#define CONDITION_C row>1 && img.data[img_index - 2 * img.step]
#define CONDITION_D col+1<img.cols && row>1 && img.data[img_index - 2 * img.step + 1]
#define CONDITION_E col+2<img.cols && row>1 && img.data[img_index - 2 * img.step + 2]

#define CONDITION_G col>1 && row>0 && img.data[img_index - img.step - 2]
#define CONDITION_H col>0 && row>0 && img.data[img_index - img.step - 1]
#define CONDITION_I row>0 && img.data[img_index - img.step]
#define CONDITION_J col+1<img.step && row>0 && img.data[img_index - img.step + 1]
#define CONDITION_K col+2<img.step && row>0 && img.data[img_index - img.step + 2]

#define CONDITION_M col>1 && img.data[img_index - 2]
#define CONDITION_N col>0 && img.data[img_index - 1]
#define CONDITION_O img.data[img_index]
#define CONDITION_P col+1<img.step && img.data[img_index + 1]

#define CONDITION_R col>0 && row+1<img.rows && img.data[img_index + img.step - 1]
#define CONDITION_S row+1<img.rows && img.data[img_index + img.step]
#define CONDITION_T col+1<img.cols && row+1<img.rows && img.data[img_index + img.step + 1]

			// Action 1: No action
#define ACTION_1  
//			// Action 2: New label (the block has foreground pixels and is not connected to anything else)
#define ACTION_2  
			//Action P: Merge with block P
#define ACTION_3 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) - 2); 
			// Action Q: Merge with block Q
#define ACTION_4 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size));	
			// Action R: Merge with block R
#define ACTION_5 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) + 2); 
			// Action S: Merge with block S
#define ACTION_6 Union(labels.data, labels_index, labels_index - 2);  
			// Action 7: Merge labels of block P and Q
#define ACTION_7 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) - 2); \
			Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size));			
			//Action 8: Merge labels of block P and R
#define ACTION_8 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) - 2); \
			Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) + 2);			
			// Action 9 Merge labels of block P and S
#define ACTION_9 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) - 2); \
			Union(labels.data, labels_index, labels_index - 2);			
			// Action 10 Merge labels of block Q and R
#define ACTION_10 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size)); \
			Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) + 2);			
			// Action 11: Merge labels of block Q and S
#define ACTION_11 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size)); \
			Union(labels.data, labels_index, labels_index - 2);			
			// Action 12: Merge labels of block R and S
#define ACTION_12 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) + 2); \
			Union(labels.data, labels_index, labels_index - 2);			
			// Action 13: not used
#define ACTION_13 
			// Action 14: Merge labels of block P, Q and S
#define ACTION_14 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) - 2); \
			Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size)); \
			Union(labels.data, labels_index, labels_index - 2);		
			//Action 15: Merge labels of block P, R and S
#define ACTION_15 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) - 2); \
			Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) + 2); \
            Union(labels.data, labels_index, labels_index - 2);			
			//Action 16: labels of block Q, R and S
#define ACTION_16 Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size)); \
			Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) + 2); \
			Union(labels.data, labels_index, labels_index - 2);			

#include "labeling_bolelli_2018_drag.inc.h"

#undef ACTION_0
#undef ACTION_2
#undef ACTION_P
#undef ACTION_Q
#undef ACTION_R
#undef ACTION_S
#undef ACTION_7
#undef ACTION_8
#undef ACTION_9
#undef ACTION_10
#undef ACTION_11
#undef ACTION_12
#undef ACTION_13
#undef ACTION_14
#undef ACTION_15
#undef ACTION_16


#undef CONDITION_B
#undef CONDITION_C
#undef CONDITION_D
#undef CONDITION_E

#undef CONDITION_G
#undef CONDITION_H
#undef CONDITION_I
#undef CONDITION_J
#undef CONDITION_K

#undef CONDITION_M
#undef CONDITION_N
#undef CONDITION_O
#undef CONDITION_P

#undef CONDITION_R
#undef CONDITION_S
#undef CONDITION_T

		}
	}

	__global__ void Compression(cuda::PtrStepSzi labels) {

		unsigned row = (blockIdx.y * BLOCK_ROWS + threadIdx.y) * 2;
		unsigned col = (blockIdx.x * BLOCK_COLS + threadIdx.x) * 2;
		unsigned labels_index = row * (labels.step / labels.elem_size) + col;

		if (row < labels.rows && col < labels.cols) {
			labels[labels_index] = Find(labels.data, labels_index);
		}
	}


	__global__ void FinalLabeling(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

		unsigned row = (blockIdx.y * BLOCK_ROWS + threadIdx.y) * 2;
		unsigned col = (blockIdx.x * BLOCK_COLS + threadIdx.x) * 2;
		unsigned labels_index = row * (labels.step / labels.elem_size) + col;
		unsigned img_index = row * (img.step / img.elem_size) + col;

		if (row < labels.rows && col < labels.cols) {

			unsigned int label = labels[labels_index] + 1;

			if (img.data[img_index])
				labels[labels_index] = label;
			else {
				labels[labels_index] = 0;
			}

			if (col + 1 < labels.cols) {
				if (img.data[img_index + 1])
					labels[labels_index + 1] = label;
				else {
					labels[labels_index + 1] = 0;
				}

				if (row + 1 < labels.rows) {
					if (img.data[img_index + img.step + 1])
						labels[labels_index + (labels.step / labels.elem_size) + 1] = label;
					else {
						labels[labels_index + (labels.step / labels.elem_size) + 1] = 0;
					}
				}
			}

			if (row + 1 < labels.rows) {
				if (img.data[img_index + img.step])
					labels[labels_index + (labels.step / labels.elem_size)] = label;
				else {
					labels[labels_index + (labels.step / labels.elem_size)] = 0;
				}
			}

		}

	}

}

class C_DRAG : public GpuLabeling2D<Connectivity2D::CONN_8> {
private:
	dim3 grid_size_;
	dim3 block_size_;

public:
	C_DRAG() {}

	void PerformLabeling() {

		d_img_labels_.create(d_img_.size(), CV_32SC1);

		grid_size_ = dim3((((d_img_.cols + 1) / 2) + BLOCK_COLS - 1) / BLOCK_COLS, (((d_img_.rows + 1) / 2) + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
		block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);

		InitLabeling << <grid_size_, block_size_ >> > (d_img_labels_);

		//cuda::GpuMat d_expanded_connections;
		//d_expanded_connections.create(d_connections_.rows * 3, d_connections_.cols * 3, CV_8UC1);
		//ExpandConnections << <grid_size_, block_size_ >> > (d_connections_, d_expanded_connections);
		//Mat1b expanded_connections;
		//d_expanded_connections.download(expanded_connections);
		//d_expanded_connections.release();

		//Mat1i init_labels;
		//d_block_labels_.download(init_labels);

		Merge << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);

		//Mat1i block_info_final;
		//d_img_labels_.download(block_info_final);		

		Compression << <grid_size_, block_size_ >> > (d_img_labels_);

		FinalLabeling << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);

		// d_img_labels_.download(img_labels_);
		hipDeviceSynchronize();
	}

	void PerformLabelingBlocksize(int x, int y, int z) override {

		d_img_labels_.create(d_img_.size(), CV_32SC1);

		grid_size_ = dim3((((d_img_.cols + 1) / 2) + x - 1) / x, (((d_img_.rows + 1) / 2) + y - 1) / y, 1);
		block_size_ = dim3(x, y, 1);

		BLOCKSIZE_KERNEL(InitLabeling, grid_size_, block_size_, 0, d_img_labels_)

		BLOCKSIZE_KERNEL(Merge, grid_size_, block_size_, 0, d_img_, d_img_labels_)

		BLOCKSIZE_KERNEL(Compression, grid_size_, block_size_, 0, d_img_labels_)

		BLOCKSIZE_KERNEL(FinalLabeling, grid_size_, block_size_, 0, d_img_, d_img_labels_)
	}


private:
	void Alloc() {
		d_img_labels_.create(d_img_.size(), CV_32SC1);
	}

	void Dealloc() {
	}

	double MemoryTransferHostToDevice() {
		perf_.start();
		d_img_.upload(img_);
		perf_.stop();
		return perf_.last();
	}

	void MemoryTransferDeviceToHost() {
		d_img_labels_.download(img_labels_);
	}

	void AllScans() {
		grid_size_ = dim3((((d_img_.cols + 1) / 2) + BLOCK_COLS - 1) / BLOCK_COLS, (((d_img_.rows + 1) / 2) + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
		block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);

		InitLabeling << <grid_size_, block_size_ >> > (d_img_labels_);

		//cuda::GpuMat d_expanded_connections;
		//d_expanded_connections.create(d_connections_.rows * 3, d_connections_.cols * 3, CV_8UC1);
		//ExpandConnections << <grid_size_, block_size_ >> > (d_connections_, d_expanded_connections);
		//Mat1b expanded_connections;
		//d_expanded_connections.download(expanded_connections);
		//d_expanded_connections.release();

		//Mat1i init_labels;
		//d_block_labels_.download(init_labels);

		Merge << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);

		//Mat1i block_info_final;
		//d_img_labels_.download(block_info_final);		

		Compression << <grid_size_, block_size_ >> > (d_img_labels_);

		FinalLabeling << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);

		hipDeviceSynchronize();
	}

public:
	void PerformLabelingWithSteps()
	{
		perf_.start();
		Alloc();
		perf_.stop();
		double alloc_timing = perf_.last();

		perf_.start();
		AllScans();
		perf_.stop();
		perf_.store(Step(StepType::ALL_SCANS), perf_.last());

		perf_.start();
		Dealloc();
		perf_.stop();
		double dealloc_timing = perf_.last();

		perf_.store(Step(StepType::ALLOC_DEALLOC), alloc_timing + dealloc_timing);
	}

};

REGISTER_LABELING(C_DRAG);

REGISTER_KERNELS(C_DRAG, InitLabeling, Merge, Compression, FinalLabeling)