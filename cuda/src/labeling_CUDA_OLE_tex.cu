#include "hip/hip_runtime.h"
// Copyright (c) 2020, the YACCLAB contributors, as 
// shown by the AUTHORS file. All rights reserved.
//
// Use of this source code is governed by a BSD-style
// license that can be found in the LICENSE file.

#include <opencv2/cudafeatures2d.hpp>

#include "hip/hip_runtime.h"
#include ""

#include "labeling_algorithms.h"
#include "register.h"


// Optimized Label Equivalence (OLE), enhanced by the use of texture memory as hinted in Asad2019


#define BLOCK_ROWS 16
#define BLOCK_COLS 16

using namespace cv;

namespace {

	// Init phase.
	// Labels start at value 1, to differentiate them from background, that has value 0.
	__global__ void Init(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

		unsigned global_row = blockIdx.y * BLOCK_ROWS + threadIdx.y;
		unsigned global_col = blockIdx.x * BLOCK_COLS + threadIdx.x;
		unsigned img_index = global_row * img.step + global_col;
		unsigned labels_index = global_row * (labels.step / labels.elem_size) + global_col;

		if (global_row < img.rows && global_col < img.cols) {
			labels[labels_index] = img[img_index] ? (labels_index + 1) : 0;
		}
	}


	__device__ unsigned int MinLabel(unsigned l1, unsigned l2) {
		if (l1 && l2)
			return min(l1, l2);
		else
			return l1;
	}


	__device__ unsigned int FindMinLabel(hipTextureObject_t texObject, unsigned row, unsigned col, unsigned label) {

		unsigned int min = label;

		min = MinLabel(min, tex2D<unsigned int>(texObject, col - 1, row - 1));
		min = MinLabel(min, tex2D<unsigned int>(texObject, col + 0, row - 1));
		min = MinLabel(min, tex2D<unsigned int>(texObject, col + 1, row - 1));
		min = MinLabel(min, tex2D<unsigned int>(texObject, col - 1, row + 0));
		min = MinLabel(min, tex2D<unsigned int>(texObject, col + 1, row + 0));
		min = MinLabel(min, tex2D<unsigned int>(texObject, col - 1, row + 1));
		min = MinLabel(min, tex2D<unsigned int>(texObject, col + 0, row + 1));
		min = MinLabel(min, tex2D<unsigned int>(texObject, col + 1, row + 1));

		return min;
	}


	// Scan phase.
	// The pixel associated with current thread is given the minimum label of the neighbours.
	__global__ void Scan(cuda::PtrStepSzi labels, hipTextureObject_t texObject, char *changes) {

		unsigned row = blockIdx.y * BLOCK_ROWS + threadIdx.y;
		unsigned col = blockIdx.x * BLOCK_COLS + threadIdx.x;
		// unsigned labels_index = row * (labels.step / labels.elem_size) + col;

		unsigned label = tex2D<unsigned int>(texObject, col, row);

		if (label) {
			unsigned min_label = FindMinLabel(texObject, row, col, label);
			if (min_label < label) {
				labels[label - 1] = min(static_cast<unsigned int>(labels[label - 1]), min_label);
				*changes = 1;
			}
		}
		
	}


	// Analysis phase.
	// The pixel associated with current thread is given the minimum label of the neighbours.
	__global__ void Analyze(cuda::PtrStepSzi labels) {

		unsigned row = blockIdx.y * BLOCK_ROWS + threadIdx.y;
		unsigned col = blockIdx.x * BLOCK_COLS + threadIdx.x;
		unsigned labels_index = row * (labels.step / labels.elem_size) + col;

		if (row < labels.rows && col < labels.cols) {

			unsigned label = labels[labels_index];

			if (label) {								

				unsigned index = labels_index;

				while (label - 1 != index) {
					index = label - 1;
					label = labels[index];
				}				

				labels[labels_index] = label;
			}
		}
	}

	__device__ unsigned int FindMinLabelNotTex(cuda::PtrStepSzi labels, unsigned row, unsigned col, unsigned label, unsigned labels_index) {

		unsigned int min = label;

		if (row > 0) {
			min = MinLabel(min, labels.data[labels_index - (labels.step / labels.elem_size)]);
			if (col > 0)
				min = MinLabel(min, labels.data[labels_index - (labels.step / labels.elem_size) - 1]);
			if (col < labels.cols - 1)
				min = MinLabel(min, labels.data[labels_index - (labels.step / labels.elem_size) + 1]);
		}
		if (row < labels.rows - 1) {
			min = MinLabel(min, labels.data[labels_index + (labels.step / labels.elem_size)]);
			if (col > 0)
				min = MinLabel(min, labels.data[labels_index + (labels.step / labels.elem_size) - 1]);
			if (col < labels.cols - 1)
				min = MinLabel(min, labels.data[labels_index + (labels.step / labels.elem_size) + 1]);
		}
		if (col > 0)
			min = MinLabel(min, labels.data[labels_index - 1]);
		if (col < labels.cols - 1)
			min = MinLabel(min, labels.data[labels_index + 1]);

		return min;
	}


	// Scan phase.
	// The pixel associated with current thread is given the minimum label of the neighbours.
	__global__ void ScanNotTex(cuda::PtrStepSzi labels, hipTextureObject_t texObject, char* changes) {

		unsigned row = blockIdx.y * BLOCK_ROWS + threadIdx.y;
		unsigned col = blockIdx.x * BLOCK_COLS + threadIdx.x;
		unsigned labels_index = row * (labels.step / labels.elem_size) + col;

		if (row < labels.rows && col < labels.cols) {

			unsigned label = labels[labels_index];

			if (label) {
				unsigned min_label = FindMinLabelNotTex(labels, row, col, label, labels_index);
				if (min_label < label) {
					labels[label - 1] = min(static_cast<unsigned int>(labels[label - 1]), min_label);
					*changes = 1;
				}
			}
		}
	}

}

class OLE_TEX : public GpuLabeling2D<Connectivity2D::CONN_8> {
private:
	dim3 grid_size_;
	dim3 block_size_;
	char changes;
	char *d_changes;

public:
	OLE_TEX() {}

	void PerformLabeling() {

		d_img_labels_.create(d_img_.size(), CV_32SC1);

		hipMalloc(&d_changes, sizeof(char));

		// Workaround for 1D images, necessary for sm >= 70
		//void (*scan_kernel) (cuda::PtrStepSzi, hipTextureObject_t, char*) = (d_img_.rows == 1 || d_img_.cols == 1) ? ScanNotTex : Scan;

		// Create Texture Object
		hipChannelFormatDesc chFormatDesc = hipCreateChannelDesc<unsigned int>();

		hipResourceDesc resDesc = {};
		resDesc.resType = hipResourceTypePitch2D;
		resDesc.res.pitch2D.devPtr = d_img_labels_.data;
		resDesc.res.pitch2D.desc = chFormatDesc;
		resDesc.res.pitch2D.width = d_img_.cols;
		resDesc.res.pitch2D.height = d_img_.rows;
		resDesc.res.pitch2D.pitchInBytes = d_img_labels_.step;

		hipTextureDesc texDesc = {
			{hipAddressModeBorder, hipAddressModeBorder},     // addressMode (fetches with out-of-range coordinates return 0)
			hipFilterModePoint,                                // filterMode (do not interpolate and take the nearest value)
			hipReadModeElementType,                            // readMode (do not convert to floating point, only for 8-bit and 16-bit integer formats)
			// other values are defaulted to 0
		};

		hipTextureObject_t texObject;
		hipCreateTextureObject(&texObject, &resDesc, &texDesc, nullptr);

		grid_size_ = dim3((d_img_.cols + BLOCK_COLS - 1) / BLOCK_COLS, (d_img_.rows + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
		block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);

		Init << <grid_size_, block_size_ >> >(d_img_, d_img_labels_);

		if (d_img_.rows == 1 || d_img_.cols == 1) {
			while (true) {
				changes = 0;
				hipMemcpy(d_changes, &changes, sizeof(char), hipMemcpyHostToDevice);

				ScanNotTex << <grid_size_, block_size_ >> > (d_img_labels_, texObject, d_changes);

				hipMemcpy(&changes, d_changes, sizeof(char), hipMemcpyDeviceToHost);

				if (!changes)
					break;

				Analyze << <grid_size_, block_size_ >> > (d_img_labels_);
			}
		}

		else {
			while (true) {
				changes = 0;
				hipMemcpy(d_changes, &changes, sizeof(char), hipMemcpyHostToDevice);

				Scan << <grid_size_, block_size_ >> > (d_img_labels_, texObject, d_changes);

				hipMemcpy(&changes, d_changes, sizeof(char), hipMemcpyDeviceToHost);

				if (!changes)
					break;

				Analyze << <grid_size_, block_size_ >> > (d_img_labels_);
			}
		}

		hipDestroyTextureObject(texObject);

		hipFree(d_changes);
		hipDeviceSynchronize();
	}


private:
	double Alloc() {
		perf_.start();
		d_img_labels_.create(d_img_.size(), CV_32SC1);
		hipMalloc(&d_changes, sizeof(char));
		perf_.stop();
		return perf_.last();
	}

	double Dealloc() {
		perf_.start();
		hipFree(d_changes);
		perf_.stop();
		return perf_.last();
	}

	double MemoryTransferHostToDevice() {
		perf_.start();
		d_img_.upload(img_);
		perf_.stop();
		return perf_.last();
	}

	void MemoryTransferDeviceToHost() {
		d_img_labels_.download(img_labels_);
	}

	void AllScans() {
		// Create Texture Object
		hipChannelFormatDesc chFormatDesc = hipCreateChannelDesc<unsigned int>();

		hipResourceDesc resDesc = {};
		resDesc.resType = hipResourceTypePitch2D;
		resDesc.res.pitch2D.devPtr = d_img_labels_.data;
		resDesc.res.pitch2D.desc = chFormatDesc;
		resDesc.res.pitch2D.width = d_img_.cols;
		resDesc.res.pitch2D.height = d_img_.rows;
		resDesc.res.pitch2D.pitchInBytes = d_img_labels_.step;

		hipTextureDesc texDesc = {
			{hipAddressModeBorder, hipAddressModeBorder},     // addressMode (fetches with out-of-range coordinates return 0)
			hipFilterModePoint,                                // filterMode (do not interpolate and take the nearest value)
			hipReadModeElementType,                            // readMode (do not convert to floating point, only for 8-bit and 16-bit integer formats)
			// other values are defaulted to 0
		};

		hipTextureObject_t texObject;
		hipCreateTextureObject(&texObject, &resDesc, &texDesc, nullptr);

		grid_size_ = dim3((d_img_.cols + BLOCK_COLS - 1) / BLOCK_COLS, (d_img_.rows + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
		block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);

		Init << <grid_size_, block_size_ >> >(d_img_, d_img_labels_);

		if (d_img_.rows == 1 || d_img_.cols == 1) {
			while (true) {
				changes = 0;
				hipMemcpy(d_changes, &changes, sizeof(char), hipMemcpyHostToDevice);

				ScanNotTex << <grid_size_, block_size_ >> > (d_img_labels_, texObject, d_changes);

				hipMemcpy(&changes, d_changes, sizeof(char), hipMemcpyDeviceToHost);

				if (!changes)
					break;

				Analyze << <grid_size_, block_size_ >> > (d_img_labels_);
			}
		}

		else {
			while (true) {
				changes = 0;
				hipMemcpy(d_changes, &changes, sizeof(char), hipMemcpyHostToDevice);

				Scan << <grid_size_, block_size_ >> > (d_img_labels_, texObject, d_changes);

				hipMemcpy(&changes, d_changes, sizeof(char), hipMemcpyDeviceToHost);

				if (!changes)
					break;

				Analyze << <grid_size_, block_size_ >> > (d_img_labels_);
			}
		}

		hipDestroyTextureObject(texObject);

		hipDeviceSynchronize();
	}

public:
	void PerformLabelingWithSteps()
	{
		double alloc_timing = Alloc();

		perf_.start();
		AllScans();
		perf_.stop();
		perf_.store(Step(StepType::ALL_SCANS), perf_.last());

		double dealloc_timing = Dealloc();

		perf_.store(Step(StepType::ALLOC_DEALLOC), alloc_timing + dealloc_timing);

	}

};

REGISTER_LABELING(OLE_TEX);
