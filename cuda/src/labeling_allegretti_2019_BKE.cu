#include "hip/hip_runtime.h"
// Copyright (c) 2020, the YACCLAB contributors, as 
// shown by the AUTHORS file. All rights reserved.
//
// Use of this source code is governed by a BSD-style
// license that can be found in the LICENSE file.

#include <opencv2/cudafeatures2d.hpp>

#include "hip/hip_runtime.h"
#include ""

#include "labeling_algorithms.h"
#include "register.h"


#define BLOCK_ROWS 16
#define BLOCK_COLS 16

using namespace cv;

namespace {

//
//         This is a block-based algorithm.
// Blocks are 2x2 sized, with internal pixels named as:
//                       +---+
//                       |a b|
//                       |c d|
//                       +---+   
// 
//       Neighbour blocks of block X are named as:
//                      +-+-+-+
//                      |P|Q|R|                
//                      +-+-+-+
//                      |S|X|
//                      +-+-+
//

enum class Info : unsigned char { a = 0, b = 1, c = 2, d = 3, P = 4, Q = 5, R = 6, S = 7 };

// Only use it with unsigned numeric types
template <typename T>
__device__ __forceinline__ unsigned char HasBit(T bitmap, Info pos) {
    return (bitmap >> static_cast<unsigned char>(pos)) & 1;
}

template <typename T>
__device__ __forceinline__ unsigned char HasBit(T bitmap, unsigned char pos) {
    return (bitmap >> pos) & 1;
}

// Only use it with unsigned numeric types
__device__ __forceinline__ void SetBit(unsigned char& bitmap, Info pos) {
    bitmap |= (1 << static_cast<unsigned char>(pos));
}

// Returns the root index of the UFTree
__device__ unsigned Find(const int* s_buf, unsigned n) {
    while (s_buf[n] != n) {
        n = s_buf[n];
    }
    return n;
}

__device__ unsigned FindAndCompress(int* s_buf, unsigned n) {
    unsigned id = n;
    while (s_buf[n] != n) {
        n = s_buf[n];
        s_buf[id] = n;
    }
    return n;
}

// Merges the UFTrees of a and b, linking one root to the other
__device__ void Union(int* s_buf, unsigned a, unsigned b) {

    bool done;

    do {

        a = Find(s_buf, a);
        b = Find(s_buf, b);

        if (a < b) {
            int old = atomicMin(s_buf + b, a);
            done = (old == b);
            b = old;
        }
        else if (b < a) {
            int old = atomicMin(s_buf + a, b);
            done = (old == a);
            a = old;
        }
        else {
            done = true;
        }

    } while (!done);

}


__global__ void InitLabeling(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels, unsigned char* last_pixel) {
    unsigned row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    unsigned col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    unsigned img_index = row * img.step + col;
    unsigned labels_index = row * (labels.step / labels.elem_size) + col;

    if (row < labels.rows && col < labels.cols) {

        unsigned P = 0;

        // Bitmask representing two kinds of information
        // Bits 0, 1, 2, 3 are set if pixel a, b, c, d are foreground, respectively
        // Bits 4, 5, 6, 7 are set if block P, Q, R, S need to be merged to X in Merge phase
        unsigned char info = 0;

        char buffer alignas(int)[4];
        *(reinterpret_cast<int*>(buffer)) = 0;

        // Read pairs of consecutive values in memory at once
        if (col + 1 < img.cols) {
            // This does not depend on endianness
            *(reinterpret_cast<int16_t*>(buffer)) = *(reinterpret_cast<int16_t*>(img.data + img_index));

            if (row + 1 < img.rows) {
                *(reinterpret_cast<int16_t*>(buffer + 2)) = *(reinterpret_cast<int16_t*>(img.data + img_index + img.step));
            }
        }
        else {
            buffer[0] = img.data[img_index];

            if (row + 1 < img.rows) {
                buffer[2] = img.data[img_index + img.step];
            }
        }

        if (buffer[0]) {
            P |= 0x777;
            SetBit(info, Info::a);
        }
        if (buffer[1]) {
            P |= (0x777 << 1);
            SetBit(info, Info::b);
        }
        if (buffer[2]) {
            P |= (0x777 << 4);
            SetBit(info, Info::c);
        }
        if (buffer[3]) {
            SetBit(info, Info::d);
        }

        if (col == 0) {
            P &= 0xEEEE;
        }
        if (col + 1 >= img.cols) {
            P &= 0x3333;
        }
        else if (col + 2 >= img.cols) {
            P &= 0x7777;
        }

        if (row == 0) {
            P &= 0xFFF0;
        }
        if (row + 1 >= img.rows) {
            P &= 0x00FF;
        }
        else if (row + 2 >= img.rows) {
            P &= 0x0FFF;
        }

        // P is now ready to be used to find neighbour blocks
        // P value avoids range errors

        int father_offset = 0;

        // P square
        if (HasBit(P, 0) && img.data[img_index - img.step - 1]) {
            father_offset = -(2 * (labels.step / labels.elem_size) + 2);
        }

        // Q square
        if ((HasBit(P, 1) && img.data[img_index - img.step]) || (HasBit(P, 2) && img.data[img_index + 1 - img.step])) {
            if (!father_offset) {
                father_offset = -(2 * (labels.step / labels.elem_size));
            }
            else {
                SetBit(info, Info::Q);
            }
        }

        // R square
        if (HasBit(P, 3) && img.data[img_index + 2 - img.step]) {
            if (!father_offset) {
                father_offset = -(2 * (labels.step / labels.elem_size) - 2);
            }
            else {
                SetBit(info, Info::R);
            }
        }

        // S square
        if ((HasBit(P, 4) && img.data[img_index - 1]) || (HasBit(P, 8) && img.data[img_index + img.step - 1])) {
            if (!father_offset) {
                father_offset = -2;
            }
            else {
                SetBit(info, Info::S);
            }
        }

        labels.data[labels_index] = labels_index + father_offset;
        if (col + 1 < labels.cols) {
            last_pixel = reinterpret_cast<unsigned char*>(labels.data + labels_index + 1);
        }
        else if (row + 1 < labels.rows) {
            last_pixel = reinterpret_cast<unsigned char*>(labels.data + labels_index + labels.step / labels.elem_size);
        }
        *last_pixel = info;
    }
}

__global__ void Merge(cuda::PtrStepSzi labels, unsigned char* last_pixel) {

    unsigned row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    unsigned col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    unsigned labels_index = row * (labels.step / labels.elem_size) + col;

    if (row < labels.rows && col < labels.cols) {

        if (col + 1 < labels.cols) {
            last_pixel = reinterpret_cast<unsigned char*>(labels.data + labels_index + 1);
        }
        else if (row + 1 < labels.rows) {
            last_pixel = reinterpret_cast<unsigned char*>(labels.data + labels_index + labels.step / labels.elem_size);
        }
        unsigned char info = *last_pixel;

        if (HasBit(info, Info::Q)) {
            Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size));
        }
        if (HasBit(info, Info::R)) {
            Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) + 2);
        }
        if (HasBit(info, Info::S)) {
            Union(labels.data, labels_index, labels_index - 2);
        }
    }
}

__global__ void Compression(cuda::PtrStepSzi labels) {
    unsigned row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    unsigned col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    unsigned labels_index = row * (labels.step / labels.elem_size) + col;

    if (row < labels.rows && col < labels.cols) {
        FindAndCompress(labels.data, labels_index);
    }
}

__global__ void FinalLabeling(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

    unsigned row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    unsigned col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    unsigned labels_index = row * (labels.step / labels.elem_size) + col;

    if (row < labels.rows && col < labels.cols) {

        int label;
        unsigned char info;
        unsigned long long buffer;

        if (col + 1 < labels.cols) {
            buffer = *reinterpret_cast<unsigned long long*>(labels.data + labels_index);
            label = (buffer & (0xFFFFFFFF)) + 1;
            info = (buffer >> 32) & 0xFFFFFFFF;
        }
        else {
            label = labels[labels_index] + 1;
            if (row + 1 < labels.rows) {
                info = labels[labels_index + labels.step / labels.elem_size];
            }
            else {
                // Read from the input image
                // "a" is already in position 0
                info = img[row * img.step + col];
            }
        }

        if (col + 1 < labels.cols) {
            *reinterpret_cast<unsigned long long*>(labels.data + labels_index) =
                (static_cast<unsigned long long>(HasBit(info, Info::b) * label) << 32) | (HasBit(info, Info::a) * label);

            if (row + 1 < labels.rows) {
                *reinterpret_cast<unsigned long long*>(labels.data + labels_index + labels.step / labels.elem_size) =
                    (static_cast<unsigned long long>(HasBit(info, Info::d) * label) << 32) | (HasBit(info, Info::c) * label);
            }
        }
        else {
            labels[labels_index] = HasBit(info, Info::a) * label;

            if (row + 1 < labels.rows) {
                labels[labels_index + (labels.step / labels.elem_size)] = HasBit(info, Info::c) * label;
            }
        }

    }

}

}

class BKE : public GpuLabeling2D<Connectivity2D::CONN_8> {
private:
    dim3 grid_size_;
    dim3 block_size_;
    unsigned char* last_pixel_;
    bool last_pixel_allocated_;

public:
    BKE() {}

    void PerformLabeling() {

        d_img_labels_.create(d_img_.size(), CV_32SC1);

        last_pixel_allocated_ = false;
        if ((d_img_.rows == 1 || d_img_.cols == 1) && !((d_img_.rows + d_img_.cols) % 2)) {
            hipMalloc(&last_pixel_, sizeof(unsigned char));
            last_pixel_allocated_ = true;
        }
        else {
            last_pixel_ = d_img_labels_.data + ((d_img_labels_.rows - 2) * d_img_labels_.step) + (d_img_labels_.cols - 2) * d_img_labels_.elemSize();
        }

        grid_size_ = dim3((((d_img_.cols + 1) / 2) + BLOCK_COLS - 1) / BLOCK_COLS, (((d_img_.rows + 1) / 2) + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
        block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);

        InitLabeling << <grid_size_, block_size_ >> > (d_img_, d_img_labels_, last_pixel_);

        //Mat1i init_blocks;
        //d_img_labels_.download(init_blocks);

        //cuda::GpuMat d_init_labels = d_img_labels_.clone();
        //FinalLabeling << <grid_size_, block_size_ >> > (d_img_, d_init_labels);
        //Mat1i init_labels;
        //d_init_labels.download(init_labels);
        //d_init_labels.release();

        Compression << <grid_size_, block_size_ >> > (d_img_labels_);

        //Mat1i compr_blocks;
        //d_img_labels_.download(compr_blocks);

        //cuda::GpuMat d_compr_labels = d_img_labels_.clone();
        //FinalLabeling << <grid_size_, block_size_ >> > (d_img_, d_compr_labels);
        //Mat1i compr_labels;
        //d_compr_labels.download(compr_labels);
        //d_compr_labels.release();

        //cuda::GpuMat d_expanded_connections;
        //d_expanded_connections.create(d_connections_.rows * 3, d_connections_.cols * 3, CV_8UC1);
        //ExpandConnections << <grid_size_, block_size_ >> > (d_connections_, d_expanded_connections);
        //Mat1b expanded_connections;
        //d_expanded_connections.download(expanded_connections);
        //d_expanded_connections.release();

        Merge << <grid_size_, block_size_ >> > (d_img_labels_, last_pixel_);

        //Mat1i merge_blocks;
        //d_img_labels_.download(merge_blocks);		

        //cuda::GpuMat d_merge_labels = d_img_labels_.clone();
        //FinalLabeling << <grid_size_, block_size_ >> > (d_img_, d_merge_labels);
        //Mat1i merge_labels;
        //d_merge_labels.download(merge_labels);
        //d_merge_labels.release();

        Compression << <grid_size_, block_size_ >> > (d_img_labels_);

        //Mat1i final_blocks;
        //d_img_labels_.download(final_blocks);

        FinalLabeling << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);

        //d_img_labels_.download(img_labels_);
        if (last_pixel_allocated_) {
            hipFree(last_pixel_);
        }
        hipDeviceSynchronize();
    }

    void PerformLabelingBlocksize(int x, int y, int z) override {

        d_img_labels_.create(d_img_.size(), CV_32SC1);

        last_pixel_allocated_ = false;
        if ((d_img_.rows == 1 || d_img_.cols == 1) && !((d_img_.rows + d_img_.cols) % 2)) {
            hipMalloc(&last_pixel_, sizeof(unsigned char));
            last_pixel_allocated_ = true;
        }
        else {
            last_pixel_ = d_img_labels_.data + ((d_img_labels_.rows - 2) * d_img_labels_.step) + (d_img_labels_.cols - 2) * d_img_labels_.elemSize();
        }

        grid_size_ = dim3((((d_img_.cols + 1) / 2) + x - 1) / x, (((d_img_.rows + 1) / 2) + y - 1) / y, 1);
        block_size_ = dim3(x, y, 1);

        BLOCKSIZE_KERNEL(InitLabeling, grid_size_, block_size_, 0, d_img_, d_img_labels_, last_pixel_)

            BLOCKSIZE_KERNEL(Compression, grid_size_, block_size_, 0, d_img_labels_)

            BLOCKSIZE_KERNEL(Merge, grid_size_, block_size_, 0, d_img_labels_, last_pixel_)

            BLOCKSIZE_KERNEL(Compression, grid_size_, block_size_, 0, d_img_labels_)

            BLOCKSIZE_KERNEL(FinalLabeling, grid_size_, block_size_, 0, d_img_, d_img_labels_)

            if (last_pixel_allocated_) {
                hipFree(last_pixel_);
            }
    }



private:
    double Alloc() {
        
        perf_.start();
        d_img_labels_.create(d_img_.size(), CV_32SC1);
        if ((d_img_.rows == 1 || d_img_.cols == 1) && !((d_img_.rows + d_img_.cols) % 2)) {
            hipMalloc(&last_pixel_, sizeof(unsigned char));
            last_pixel_allocated_ = true;
        }
        else {
            last_pixel_ = d_img_labels_.data + ((d_img_labels_.rows - 2) * d_img_labels_.step) + (d_img_labels_.cols - 2) * d_img_labels_.elemSize();
        }        
        hipMemset2D(d_img_labels_.data, d_img_labels_.step, 0, d_img_labels_.cols * 4, d_img_labels_.rows);
        if (last_pixel_allocated_) {
            hipMemset(last_pixel_, 0, 1);
        }
        hipDeviceSynchronize();
        double t = perf_.stop();

        perf_.start();
        hipMemset2D(d_img_labels_.data, d_img_labels_.step, 0, d_img_labels_.cols * 4, d_img_labels_.rows);
        if (last_pixel_allocated_) {
            hipMemset(last_pixel_, 0, 1);
        }
        hipDeviceSynchronize();

        t -= perf_.stop();
        return t;
    }

    void Dealloc() {
        if (last_pixel_allocated_) {
            hipFree(last_pixel_);
        }
    }

    double MemoryTransferHostToDevice() {
        perf_.start();
        d_img_.upload(img_);
        perf_.stop();
        return perf_.last();
    }

    void MemoryTransferDeviceToHost() {
        d_img_labels_.download(img_labels_);
    }

    void AllScans() {
        grid_size_ = dim3((((d_img_.cols + 1) / 2) + BLOCK_COLS - 1) / BLOCK_COLS, (((d_img_.rows + 1) / 2) + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
        block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);

        InitLabeling << <grid_size_, block_size_ >> > (d_img_, d_img_labels_, last_pixel_);

        Compression << <grid_size_, block_size_ >> > (d_img_labels_);

        //cuda::GpuMat d_expanded_connections;
        //d_expanded_connections.create(d_connections_.rows * 3, d_connections_.cols * 3, CV_8UC1);
        //ExpandConnections << <grid_size_, block_size_ >> > (d_connections_, d_expanded_connections);
        //Mat1b expanded_connections;
        //d_expanded_connections.download(expanded_connections);
        //d_expanded_connections.release();

        //Mat1i init_labels;
        //d_block_labels_.download(init_labels);

        Merge << <grid_size_, block_size_ >> > (d_img_labels_, last_pixel_);

        //Mat1i block_info_final;
        //d_img_labels_.download(block_info_final);		

        Compression << <grid_size_, block_size_ >> > (d_img_labels_);

        FinalLabeling << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);

        hipDeviceSynchronize();
    }

public:
    void PerformLabelingWithSteps()
    {

        double alloc_timing = Alloc();

        perf_.start();
        AllScans();
        perf_.stop();
        perf_.store(Step(StepType::ALL_SCANS), perf_.last());

        perf_.start();
        Dealloc();
        perf_.stop();
        double dealloc_timing = perf_.last();

        perf_.store(Step(StepType::ALLOC_DEALLOC), alloc_timing + dealloc_timing);
    }

};

REGISTER_LABELING(BKE)

REGISTER_KERNELS(BKE, InitLabeling, Compression, Merge, FinalLabeling)