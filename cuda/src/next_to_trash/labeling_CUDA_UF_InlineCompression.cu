#include "hip/hip_runtime.h"
// Copyright (c) 2020, the YACCLAB contributors, as 
// shown by the AUTHORS file. All rights reserved.
//
// Use of this source code is governed by a BSD-style
// license that can be found in the LICENSE file.

#include <opencv2/cudafeatures2d.hpp>

#include "hip/hip_runtime.h"
#include ""

#include "labeling_algorithms.h"
#include "register.h"

// Questo algoritmo � una modifica dell' Union Find (BUF) che esegue usa la FindAndCompress al posto della 
// find usata dall'UF. La FindAndCompress aggiorna la label del pixel di partenza ad ogni iterazione della 
// procedura di ricerca della label root. Ovvert se l'albero di equivalenza � cos� costruito: 

//       A
//     /
//    B
//   /
//  C

// allora all prima iterazione aggiorno la label di C sostituendola con B e all'iterazione successiva con A.
// In questo modo se un altro thread legge il mio valore a met� trova gi� B ed evita un passaggio. Funziona meglio 
// dell'UF. 

// Il minimo per entrambi � 4
#define BLOCK_ROWS 16
#define BLOCK_COLS 16

using namespace cv;

namespace {

    // Risale alla radice dell'albero a partire da un suo nodo n
    __device__ unsigned Find(const int *s_buf, unsigned n) {
        // Attenzione: non invocare la find su un pixel di background

        unsigned label = s_buf[n];

        assert(label > 0);

        while (label - 1 != n) {
            n = label - 1;
            label = s_buf[n];

            assert(label > 0);
        }

        return n;

    }

    // Risale alla radice dell'albero a partire da un suo nodo n
    __device__ unsigned FindCompress(int *s_buf, unsigned n) {
        // Attenzione: non invocare la find su un pixel di background

        unsigned id = n;

        unsigned label = s_buf[n];

        assert(label > 0);

        while (label - 1 != n) {
            n = label - 1;
            label = s_buf[n];
            s_buf[id] = label;
            assert(label > 0);
        }

        return n;
    }


    // Unisce gli alberi contenenti i nodi a e b, collegandone le radici
    __device__ void Union(int *s_buf, unsigned a, unsigned b) {

        bool done;

        do {

            a = Find(s_buf, a);
            b = Find(s_buf, b);

            if (a < b) {
                int old = atomicMin(s_buf + b, a + 1);
                done = (old == b + 1);
                b = old - 1;
            }
            else if (b < a) {
                int old = atomicMin(s_buf + a, b + 1);
                done = (old == a + 1);
                a = old - 1;
            }
            else {
                done = true;
            }

        } while (!done);

    }


    //Effettuo il controllo sui 4 vicini della maschera
    //Prova a sincronizzare dopo ogni vicino
    __global__ void LocalMerge(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

        unsigned local_row = threadIdx.y;
        unsigned local_col = threadIdx.x;
        unsigned local_index = local_row * BLOCK_COLS + local_col;

        unsigned global_row = blockIdx.y * BLOCK_ROWS + local_row;
        unsigned global_col = blockIdx.x * BLOCK_COLS + local_col;
        unsigned img_index = global_row * img.step + global_col;

        __shared__ int s_buf[BLOCK_ROWS * BLOCK_COLS];
        __shared__ unsigned char s_img[BLOCK_ROWS * BLOCK_COLS];

        bool in_limits = (global_row < img.rows && global_col < img.cols);

        s_buf[local_index] = local_index + 1;
        s_img[local_index] = in_limits ? img[img_index] : 0xFF;

        __syncthreads();

        unsigned char v = s_img[local_index];

        if (in_limits) {

            if (v) {

                if (local_col > 0 && s_img[local_index - 1]) {
                    Union(s_buf, local_index, local_index - 1);
                }


                if (local_row > 0 && s_img[local_index - BLOCK_COLS]) {
                    Union(s_buf, local_index, local_index - BLOCK_COLS);
                }

            }

            else {
                if (local_row > 0 && s_img[local_index - BLOCK_COLS]) {

                    if (local_col > 0 && s_img[local_index - 1]) {
                        Union(s_buf, local_index - 1, local_index - BLOCK_COLS);
                    }


                    if (local_col < BLOCK_COLS - 1 && s_img[local_index + 1]) {
                        Union(s_buf, local_index + 1, local_index - BLOCK_COLS);
                    }
                }

            }

        }

        __syncthreads();

        if (in_limits) {

            if (v) {
                unsigned f = FindCompress(s_buf, local_index);
                unsigned f_row = f / BLOCK_COLS;
                unsigned f_col = f % BLOCK_COLS;
                unsigned global_f = (blockIdx.y * BLOCK_ROWS + f_row) * (labels.step / labels.elem_size) + (blockIdx.x * BLOCK_COLS + f_col);
                labels.data[global_row * labels.step / sizeof(int) + global_col] = global_f + 1;		// C'� distinzione tra background e foreground
            }

            else {
                labels.data[global_row * labels.step / sizeof(int) + global_col] = 0;
            }

        }
    }


    __global__ void GlobalMerge(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

        unsigned local_row = threadIdx.y;
        unsigned local_col = threadIdx.x;

        unsigned global_row = blockIdx.y * BLOCK_ROWS + local_row;
        unsigned global_col = blockIdx.x * BLOCK_COLS + local_col;
        unsigned img_index = global_row * img.step + global_col;
        unsigned labels_index = global_row * (labels.step / labels.elem_size) + global_col;

        bool in_limits = (global_row < img.rows && global_col < img.cols);

        if (in_limits) {

            unsigned char v = img[img_index];

            if (v) {

                if (global_col > 0 && local_col == 0 && img[img_index - 1]) {
                    Union(labels.data, labels_index, labels_index - 1);
                }

                if (global_row > 0 && local_row == 0 && img[img_index - img.step]) {
                    Union(labels.data, labels_index, labels_index - labels.step / sizeof(int));
                }

            }

            else {

                if (global_row > 0 && img[img_index - img.step]) {

                    if (global_col > 0 && (local_row == 0 || local_col == 0) && img[img_index - 1]) {
                        Union(labels.data, labels_index - 1, labels_index - labels.step / sizeof(int));
                    }

                    if ((global_col < img.cols - 1) && (local_row == 0 || local_col == BLOCK_COLS - 1) && img[img_index + 1]) {
                        Union(labels.data, labels_index + 1, labels_index - labels.step / sizeof(int));
                    }
                }
            }

        }
    }


    __global__ void PathCompression(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

        unsigned global_row = blockIdx.y * BLOCK_ROWS + threadIdx.y;
        unsigned global_col = blockIdx.x * BLOCK_COLS + threadIdx.x;
        unsigned labels_index = global_row * (labels.step / labels.elem_size) + global_col;

        if (global_row < labels.rows && global_col < labels.cols) {
            unsigned char val = img[global_row * img.step + global_col];
            if (val) {
                labels[labels_index] = Find(labels.data, labels_index) + 1;
            }
        }
    }

}

class CUDA_UF_InlineCompression : public GpuLabeling2D<CONN_8> {
private:
    dim3 grid_size_;
    dim3 block_size_;

public:
    CUDA_UF_InlineCompression() {}

    void PerformLabeling() {

        d_img_labels_.create(d_img_.size(), CV_32SC1);
        grid_size_ = dim3((d_img_.cols + BLOCK_COLS - 1) / BLOCK_COLS, (d_img_.rows + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
        block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);

        // Phase 1
        // Etichetta i pixel localmente al blocco		
        LocalMerge << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);

        // Immagine di debug della prima fase
        //cuda::GpuMat d_local_labels;
        //d_img_labels_.copyTo(d_local_labels);
        //PathCompression << <grid_size_, block_size_ >> > (d_img_, d_local_labels);
        //// ZeroBackground << <grid_size_, block_size_ >> > (d_img_, d_local_labels);
        //Mat1i local_labels(img_.size());
        //d_local_labels.download(local_labels);

        // Phase 2
        // Collega tra loro gli alberi union-find dei diversi blocchi
        GlobalMerge << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);

        // Immagine di debug della seconda fase
        //cuda::GpuMat d_global_labels;
        //d_img_labels_.copyTo(d_global_labels);
        //PathCompression << <grid_size_, block_size_ >> > (d_img_, d_global_labels);
        //// ZeroBackground << <grid_size_, block_size_ >> > (d_img_, d_global_labels);
        //Mat1i global_labels(img_.size());
        //d_global_labels.download(global_labels);

        // Phase 3
        // Collassa gli alberi union-find sulle radici
        PathCompression << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);

        hipDeviceSynchronize();
    }


private:
    double Alloc() {
        perf_.start();
        d_img_labels_.create(d_img_.size(), CV_32SC1);
        perf_.stop();
        return perf_.last();
    }

    double Dealloc() {
        perf_.start();
        perf_.stop();
        return perf_.last();
    }

    double MemoryTransferHostToDevice() {
        perf_.start();
        d_img_.upload(img_);
        perf_.stop();
        return perf_.last();
    }

    void MemoryTransferDeviceToHost() {
        d_img_labels_.download(img_labels_);
    }

    void LocalScan() {
        grid_size_ = dim3((d_img_.cols + BLOCK_COLS - 1) / BLOCK_COLS, (d_img_.rows + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
        block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);
        LocalMerge << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);
        hipDeviceSynchronize();
    }

    void GlobalScan() {
        GlobalMerge << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);
        PathCompression << <grid_size_, block_size_ >> > (d_img_, d_img_labels_);
        hipDeviceSynchronize();
    }

public:
    void PerformLabelingWithSteps()
    {
        double alloc_timing = Alloc();

        perf_.start();
        LocalScan();
        perf_.stop();
        perf_.store(Step(StepType::FIRST_SCAN), perf_.last());

        perf_.start();
        GlobalScan();
        perf_.stop();
        perf_.store(Step(StepType::SECOND_SCAN), perf_.last());

        double dealloc_timing = Dealloc();

        perf_.store(Step(StepType::ALLOC_DEALLOC), alloc_timing + dealloc_timing);

    }

};

REGISTER_LABELING(CUDA_UF_InlineCompression);

